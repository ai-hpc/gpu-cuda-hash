#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <sstream>
#include <fstream>
#include <string>
#include <chrono>
#include <hip/hip_runtime.h>
#include <vector>
#include <unordered_map>
#include <cstdint>

// Add these color definitions at the top
#define RED     "\033[31m"
#define GREEN   "\033[32m"
#define YELLOW  "\033[33m"
#define BLUE    "\033[34m"
#define MAGENTA "\033[35m"
#define CYAN    "\033[36m"
#define RESET   "\033[0m"
#define BOLD    "\033[1m"

#define MAX_FOUND 1000


__constant__ const unsigned long long total_passwords = 62ULL * 62 * 62 * 62 * 62 * 62;
__constant__ char charset[] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789";
__constant__ double reciprocal = 1.0 / 62.0;

struct FoundPassword {
    char password[7];
    uint8_t hash[32];
    uint8_t salt[8];
};

// Right rotate function
__device__ __forceinline__ uint32_t rotr(uint32_t x, uint32_t n) {
    uint32_t result;
    asm("shf.r.wrap.b32 %0, %1, %1, %2;" : "=r"(result) : "r"(x), "r"(n));
    return result;
}

// SHA-256 hash function
__device__ void sha256(const uint8_t* __restrict__ data, uint8_t* __restrict__ hash) {
    // Initial hash values
    uint32_t a = 0x6a09e667;
    uint32_t b = 0xbb67ae85;
    uint32_t c = 0x3c6ef372;
    uint32_t d = 0xa54ff53a;
    uint32_t e = 0x510e527f;
    uint32_t f = 0x9b05688c;
    uint32_t g = 0x1f83d9ab;
    uint32_t h = 0x5be0cd19;

    static const uint32_t K[64] = {
        0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
        0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
        0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
        0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
        0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
        0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
        0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
        0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
        0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
        0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
        0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
        0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
        0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
        0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
        0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
        0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

    uint32_t W[64];
    W[0] = ((uint32_t)data[0] << 24) | ((uint32_t)data[1] << 16) | ((uint32_t)data[2] << 8) | data[3];
    W[1] = ((uint32_t)data[4] << 24) | ((uint32_t)data[5] << 16) | ((uint32_t)data[6] << 8) | data[7];
    W[2] = ((uint32_t)data[8] << 24) | ((uint32_t)data[9] << 16) | ((uint32_t)data[10] << 8) | data[11];
    W[3] = ((uint32_t)data[12] << 24) | ((uint32_t)data[13] << 16) | 0x8000;

    *(uint4*)&W[4] = make_uint4(0, 0, 0, 0);
    *(uint4*)&W[8] = make_uint4(0, 0, 0, 0);
    *(uint4*)&W[12] = make_uint4(0, 0, 0, 112);

    #pragma unroll 48
    for (int i = 16; i < 64; i++) {
        uint32_t s0 = rotr(W[i - 15], 7) ^ rotr(W[i - 15], 18) ^ (W[i - 15] >> 3);
        uint32_t s1 = rotr(W[i - 2], 17) ^ rotr(W[i - 2], 19) ^ (W[i - 2] >> 10);
        W[i] = W[i - 16] + s0 + W[i - 7] + s1;
    }

    #pragma unroll 64
    for (int i = 0; i < 64; i++) {
        uint32_t S1 = rotr(e, 6) ^ rotr(e, 11) ^ rotr(e, 25);
        uint32_t ch = (e & f) ^ (~e & g);
        uint32_t temp1 = h + S1 + ch + K[i] + W[i];
        uint32_t S0 = rotr(a, 2) ^ rotr(a, 13) ^ rotr(a, 22);
        uint32_t maj = (a & b) ^ (a & c) ^ (b & c);
        uint32_t temp2 = S0 + maj;

        h = g;
        g = f;
        f = e;
        e = d + temp1;
        d = c;
        c = b;
        b = a;
        a = temp1 + temp2;
    }

    // Add the compressed chunk to the current hash value
    a += 0x6a09e667;
    b += 0xbb67ae85;
    c += 0x3c6ef372;
    d += 0xa54ff53a;
    e += 0x510e527f;
    f += 0x9b05688c;
    g += 0x1f83d9ab;
    h += 0x5be0cd19;

    // Produce the final hash value (big-endian) without using a loop
    hash[0] = a >> 24;
    hash[1] = a >> 16;
    hash[2] = a >> 8;
    hash[3] = a;

    hash[4] = b >> 24;
    hash[5] = b >> 16;
    hash[6] = b >> 8;
    hash[7] = b;

    hash[8] = c >> 24;
    hash[9] = c >> 16;
    hash[10] = c >> 8;
    hash[11] = c;

    hash[12] = d >> 24;
    hash[13] = d >> 16;
    hash[14] = d >> 8;
    hash[15] = d;

    hash[16] = e >> 24;
    hash[17] = e >> 16;
    hash[18] = e >> 8;
    hash[19] = e;

    hash[20] = f >> 24;
    hash[21] = f >> 16;
    hash[22] = f >> 8;
    hash[23] = f;

    hash[24] = g >> 24;
    hash[25] = g >> 16;
    hash[26] = g >> 8;
    hash[27] = g;

    hash[28] = h >> 24;
    hash[29] = h >> 16;
    hash[30] = h >> 8;
    hash[31] = h;

}


// Fix the hexToBytes function to maintain byte order
void hexToBytes(const char* hex, uint8_t* bytes) {
    for (int i = 0; i < strlen(hex)/2; i++) {
        sscanf(hex + i*2, "%2hhx", &bytes[i]);
    }
}

int f(const uint8_t* data, int length) {
    unsigned int hash = 0;
    for (int i = 0; i < length; ++i) {
        hash = hash * 31 + data[i];
    }
    return hash % 1999997;
}

__device__ int f2(const uint8_t* data, int length) {
    unsigned int hash = 0;
    for (int i = 0; i < length; ++i) {
        hash = hash * 31 + data[i];
    }
    return hash % 1999997;
}

// Node structure for AVL Tree
struct AVLNode {
    uint8_t hash[32];
    AVLNode* left;
    AVLNode* right;
    int height;
};

// Function to create a new AVL node
AVLNode* createNode(const uint8_t* hash) {
    AVLNode* node = new AVLNode();
    std::copy(hash, hash + 32, node->hash);
    node->left = node->right = nullptr;
    node->height = 1; // Initial height of a new node is 1
    return node;
}

// Function to get the height of the tree
int height(AVLNode* node) {
    return node ? node->height : 0;
}

// Function to get the balance factor of a node
int getBalance(AVLNode* node) {
    return node ? height(node->left) - height(node->right) : 0;
}

// Right rotate the subtree rooted with y
AVLNode* rightRotate(AVLNode* y) {
    AVLNode* x = y->left;
    AVLNode* T2 = x->right;

    // Perform rotation
    x->right = y;
    y->left = T2;

    // Update heights
    y->height = std::max(height(y->left), height(y->right)) + 1;
    x->height = std::max(height(x->left), height(x->right)) + 1;

    // Return new root
    return x;
}

// Left rotate the subtree rooted with x
AVLNode* leftRotate(AVLNode* x) {
    AVLNode* y = x->right;
    AVLNode* T2 = y->left;

    // Perform rotation
    y->left = x;
    x->right = T2;

    // Update heights
    x->height = std::max(height(x->left), height(x->right)) + 1;
    y->height = std::max(height(y->left), height(y->right)) + 1;

    // Return new root
    return y;
}

// AVL tree insertion logic
AVLNode* insert(AVLNode* node, const uint8_t* hash) {
    if (!node) return createNode(hash);

    if (std::lexicographical_compare(hash, hash + 32, node->hash, node->hash + 32)) {
        node->left = insert(node->left, hash);
    } else if (std::lexicographical_compare(node->hash, node->hash + 32, hash, hash + 32)) {
        node->right = insert(node->right, hash);
    } else {
        return node; // Duplicate hashes are not allowed
    }

    // Update height and balance the tree
    node->height = 1 + std::max(height(node->left), height(node->right));
    int balance = getBalance(node);

    // Perform rotations if necessary
    if (balance > 1 && std::lexicographical_compare(hash, hash + 32, node->left->hash, node->left->hash + 32)) {
        return rightRotate(node);
    }
    if (balance < -1 && std::lexicographical_compare(node->right->hash, node->right->hash + 32, hash, hash + 32)) {
        return leftRotate(node);
    }
    if (balance > 1 && std::lexicographical_compare(node->left->hash, node->left->hash + 32, hash, hash + 32)) {
        node->left = leftRotate(node->left);
        return rightRotate(node);
    }
    if (balance < -1 && std::lexicographical_compare(hash, hash + 32, node->right->hash, node->right->hash + 32)) {
        node->right = rightRotate(node->right);
        return leftRotate(node);
    }

    return node;
}

// In-order traversal to flatten the AVL tree
void inOrderTraversal(AVLNode* node, std::vector<std::vector<uint8_t>>& sortedHashes) {
    if (node) {
        inOrderTraversal(node->left, sortedHashes);
        sortedHashes.push_back(std::vector<uint8_t>(node->hash, node->hash + 32));
        inOrderTraversal(node->right, sortedHashes);
    }
}

// Function to insert hashes into the AVL tree
AVLNode* insertHashesIntoAVLTree(uint8_t all_target_hashes[10][100][32]) {
    AVLNode* root = nullptr;

    for (int salt_index = 0; salt_index < 10; salt_index++) {
        for (int hash_index = 0; hash_index < 100; hash_index++) {
            // Insert the hash into the AVL tree
            root = insert(root, all_target_hashes[salt_index][hash_index]);
        }
    }

    return root;
}

__device__ int compareHashes(const uint8_t* hash1, const uint8_t* hash2) {
    for (int i = 0; i < 32; ++i) {
        if (hash1[i] < hash2[i]) return -1;
        if (hash1[i] > hash2[i]) return 1;
    }
    return 0;
}

__device__ bool binarySearchHashes(const uint8_t* sortedHashes, int num_hashes, const uint8_t* targetHash) {
    int left = 0;
    int right = num_hashes - 1;

    while (left <= right) {
        int mid = left + (right - left) / 2;

        const uint8_t* midHash = &sortedHashes[mid * 32];
        int cmp = compareHashes(targetHash, midHash);

        if (cmp == 0) {
            return true; // Match found
        } else if (cmp < 0) {
            right = mid - 1;
        } else {
            left = mid + 1;
        }
    }
    return false; // No match found
}

__global__ void find_passwords_optimized_multi(
    const uint8_t* __restrict__ target_salts,
    const uint8_t* __restrict__ sortedHashes,
    const uint8_t* __restrict__ target_hashes,
    int num_hashes,
    FoundPassword* __restrict__ found_passwords,
    int* __restrict__ num_found,
    const int* __restrict__ d_hash_data,
    int hash_table_size
) {
    __shared__ uint8_t shared_salt[8];
    uint8_t hash[32];
    uint8_t combined[14];

    // Calculate thread position for parallel password generation
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Iterate over each salt
    for (int salt_idx = 0; salt_idx < 10; ++salt_idx) {
        // Load the current salt into shared memory
        if (threadIdx.x < 8) {
            shared_salt[threadIdx.x] = target_salts[salt_idx * 8 + threadIdx.x];
        }

        __syncthreads();
        

        // Process multiple passwords per thread using stride
        for (uint64_t password_idx = tid; password_idx < total_passwords; password_idx += 1572864) {
            uint64_t idx = password_idx;

            
            
            #pragma unroll
            for (int i = 0; i < 6; ++i) {
                combined[i] = charset[idx % 62];
                combined[6 + i] = shared_salt[i];
                idx = static_cast<uint64_t>(idx * reciprocal); // Approximate division by 62
            }       
            // Use shared memory for salt
            combined[12] = shared_salt[6];
            combined[13] = shared_salt[7];
            
            sha256(combined, hash);
            int index = f2(hash, 8);

            // Use binary search to find the hash
            // if (binarySearchHashes(sortedHashes, num_hashes, hash)) {
            //     atomicAdd(num_found, 1);
            //     return; // Early exit for this thread
            // }
            
            // Use linear probing to resolve collisions
            while (d_hash_data[index] != -1) {

                // int target_index = d_hash_data[index];
                // const uint8_t* current_target = &target_hashes[target_index * 32];
                
                if (binarySearchHashes(sortedHashes, num_hashes, hash)) {
                    atomicAdd(num_found, 1);
                    return; // Early exit for this thread
                }
                // bool match = true;
                // #pragma unroll 8
                // for (int k = 28; k < 32; k += 4) {
                //     if (*(uint32_t*)&hash[k] != *(uint32_t*)&current_target[k]) {
                //         match = false;
                //         break;
                //     }
                // }
                // if (match) {
                //     atomicAdd(num_found, 1);
                //     // printf("index: %d, hash_data[index]: %d\n", index, d_hash_data[index]);
                //     break;
                //     // printf("Found password: %s\n", combined);
                //     // int found_idx = atomicAdd(num_found, 1);
                // }
                index += 1;
            }
            //     // Get the target hash index from the hash table
            //     int target_index = d_hash_data[index];
            //     const uint8_t* current_target = &target_hashes[target_index * 32];
            
            //     // Compare the computed hash with the target hash
            //     bool match = true;
            //     #pragma unroll 8
            //     for (int k = 0; k < 32; k += 4) {
            //         if (*(uint32_t*)&hash[k] != *(uint32_t*)&current_target[k]) {
            //             match = false;
            //             break;
            //         }
            //     }
            
            //     if (match) {
            //         int found_idx = atomicAdd(num_found, 1);
            //         if (found_idx < MAX_FOUND) {
            //             // Directly assign characters to the password array
            //             found_passwords[found_idx].password[0] = combined[0];
            //             found_passwords[found_idx].password[1] = combined[1];
            //             found_passwords[found_idx].password[2] = combined[2];
            //             found_passwords[found_idx].password[3] = combined[3];
            //             found_passwords[found_idx].password[4] = combined[4];
            //             found_passwords[found_idx].password[5] = combined[5];
            //             found_passwords[found_idx].password[6] = '\0'; // Null-terminate the string

            //             // Use a loop to copy the hash and salt, which are larger
            //             #pragma unroll
            //             for (int i = 0; i < 32; ++i) {
            //                 found_passwords[found_idx].hash[i] = hash[i];
            //             }

            //             #pragma unroll
            //             for (int i = 0; i < 8; ++i) {
            //                 found_passwords[found_idx].salt[i] = shared_salt[i];
            //             }
            //         }
            //         break; // Exit loop once a match is found
            //     }
            
            //     // Move to the next index in case of a collision
            //     index = index + 1;
            // }
        }
    }
}





int main() {

    int numDevices;
    hipGetDeviceCount(&numDevices);

    if (numDevices < 1) {
        std::cerr << "No CUDA-capable devices found." << std::endl;
        return 1;
    } else if (numDevices == 1) {
        std::cout << "Using device: " << 0 << std::endl;
    }
    // Get device properties
    int maxThreadsPerBlock, maxBlocksPerSM, numSMs;
    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0);
    hipDeviceGetAttribute(&maxBlocksPerSM, hipDeviceAttributeMaxBlocksPerMultiprocessor, 0);
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);

    // printf("Device properties:\n");
    // printf("- Number of SMs: %d\n", numSMs);
    // printf("- Max threads per block: %d\n", maxThreadsPerBlock);
    // printf("- Max blocks per SM: %d\n", maxBlocksPerSM);

    uint8_t all_target_hashes[10][100][32]; // 10 salts, each with 100 hashes
    uint8_t all_target_salts[10][8];        // 10 unique salts

    std::ifstream infile("in.txt");
    if (!infile) {
        printf("Error: Unable to open file in.txt\n");
        return 1;
    }

    std::string line;
    int salt_index = 0;
    int hash_index = 0;
    while (std::getline(infile, line) && salt_index < 10) {
        // Convert the hash from hex to bytes and store it
        hexToBytes(line.substr(0, 64).c_str(), all_target_hashes[salt_index][hash_index]);

        // Store the salt only once for each group of 100 hashes
        if (hash_index == 0) {
            hexToBytes(line.substr(65, 16).c_str(), all_target_salts[salt_index]);
        }

        hash_index++;
        if (hash_index >= 100) {
            hash_index = 0;
            salt_index++;
        }
    }

    AVLNode* root = insertHashesIntoAVLTree(all_target_hashes);
    
    // Flatten the AVL tree into a sorted array
    std::vector<std::vector<uint8_t>> sortedHashes;
    inOrderTraversal(root, sortedHashes);

    // Allocate and copy sorted hashes to device
    uint8_t* d_sortedHashes;
    hipMalloc(&d_sortedHashes, sortedHashes.size() * 32 * sizeof(uint8_t));
    for (size_t i = 0; i < sortedHashes.size(); ++i) {
        hipMemcpy(d_sortedHashes + i * 32, sortedHashes[i].data(), 32 * sizeof(uint8_t), hipMemcpyHostToDevice);
    }


    const int HASH_TABLE_SIZE = 1999997; // Adjusted to accommodate 1000 target hashes

    // Initialize and populate hash table
    std::vector<int> hash_data(HASH_TABLE_SIZE, -1);

    for (int salt_index = 0; salt_index < 10; salt_index++) {
        for (int hash_index = 0; hash_index < 100; hash_index++) {
            // Calculate the hash value for the current hash
            int index = f(all_target_hashes[salt_index][hash_index], 8);

            // Use linear probing to resolve collisions
            while (hash_data[index] != -1) {
                index = (index + 1) % HASH_TABLE_SIZE;
            }

            // Store the index of the hash in the hash table
            hash_data[index] = salt_index * 100 + hash_index;

            // printf("index: %d, hash_data[index]: %d\n", index, hash_data[index]);
        }
    }

    // Declare device pointers
    uint8_t *d_target_salts;
    uint8_t *d_target_hashes;

    // Allocate memory for 10 salts, each 8 bytes
    hipMalloc(&d_target_salts, 10 * 8 * sizeof(uint8_t));

    // Allocate memory for 1000 hashes, each 32 bytes
    hipMalloc(&d_target_hashes, 1000 * 32 * sizeof(uint8_t));


    // Copy 10 salts, each 8 bytes, from host to device
    hipMemcpy(d_target_salts, all_target_salts, 10 * 8 * sizeof(uint8_t), hipMemcpyHostToDevice);

    // Copy 1000 hashes, each 32 bytes, from host to device
    hipMemcpy(d_target_hashes, all_target_hashes, 1000 * 32 * sizeof(uint8_t), hipMemcpyHostToDevice);
    
    // Allocate memory for the hash table on the device
    int* d_hash_data;
    hipMalloc(&d_hash_data, HASH_TABLE_SIZE * sizeof(int));

    // Copy the initialized hash table from host to device
    hipMemcpy(d_hash_data, hash_data.data(), HASH_TABLE_SIZE * sizeof(int), hipMemcpyHostToDevice);


    // Determine the number of threads per block
    int blockSize = 512; // Choose a block size that is a multiple of the warp size

    // Calculate the total number of threads needed
    uint64_t totalThreads = total_passwords;

    // Calculate the number of blocks needed to cover all threads
    int numBlocks = (totalThreads + blockSize - 1) / blockSize;

    // Ensure the number of blocks does not exceed the maximum allowed by the device
    numBlocks = min(numBlocks, numSMs * maxBlocksPerSM);

    // printf("Kernel configuration:\n");
    // printf("- Block size: %d\n", blockSize);
    // printf("- Number of blocks: %d\n", numBlocks);

    // Allocate memory for found passwords on the device
    FoundPassword* d_found_passwords;
    hipMalloc(&d_found_passwords, MAX_FOUND * sizeof(FoundPassword));

    // Allocate memory for the number of found passwords on the device
    int* d_num_found;
    hipMalloc(&d_num_found, sizeof(int));

    // Initialize the number of found passwords to zero
    hipMemset(d_num_found, 0, sizeof(int));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    find_passwords_optimized_multi<<<numBlocks, blockSize>>>(
        d_target_salts,       // Device pointer to the array of salts
        d_sortedHashes,
        d_target_hashes,      // Device pointer to the array of hashes
        1000,                 // Total number of hashes (10 salts * 100 hashes each)
        d_found_passwords,    // Device pointer to store found passwords
        d_num_found,          // Device pointer to store the number of found passwords
        d_hash_data,          // Device pointer to the hash table data
        HASH_TABLE_SIZE       // Size of the hash table
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float gpu_time_ms;
    hipEventElapsedTime(&gpu_time_ms, start, stop);
    
    hipDeviceSynchronize();

    // Allocate memory on the host to store found passwords
    FoundPassword* h_found_passwords = new FoundPassword[MAX_FOUND];

    // Variable to store the number of found passwords
    int h_num_found;

    // Copy the number of found passwords from device to host
    hipMemcpy(&h_num_found, d_num_found, sizeof(int), hipMemcpyDeviceToHost);

    // Copy the found passwords from device to host
    hipMemcpy(h_found_passwords, d_found_passwords, h_num_found * sizeof(FoundPassword), hipMemcpyDeviceToHost);

    // Iterate over the found passwords and print their details
    // for (int i = 0; i < h_num_found; i++) {
    //     const FoundPassword& fp = h_found_passwords[i];
        
    //     // Print the hash
    //     for (int j = 0; j < 32; j++) {
    //         printf("%02x", fp.hash[j]);
    //     }
    //     printf(":");
        
    //     // Print the salt
    //     for (int j = 0; j < 8; j++) {
    //         printf("%02x", fp.salt[j]);
    //     }
    //     printf(":%s\n", fp.password);
    // }

    // Print the total number of found passwords
    printf("\nFound %d passwords\n", h_num_found);



    printf(BOLD CYAN "\nPerformance Metrics:\n" RESET);
    printf("GPU Time: %.2f ms\n", gpu_time_ms);
    // printf("Performance: %.2f GH/s\n", total_passwords / elapsed_seconds.count() / 1e9);


    hipFree(d_found_passwords);
    hipFree(d_num_found);
    hipFree(d_target_salts);
    hipFree(d_target_hashes);
    hipFree(d_hash_data);
    hipFree(d_sortedHashes);

    return 0;
}